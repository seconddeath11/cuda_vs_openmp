
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16


__global__ void matrix_mult(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 



int main(int argc, char const *argv[])
{
    int m, n, k;
    srand(3333);
    m = 9000;
    n = 9000;
    k = 9000;

    int *a, *b;
    hipHostMalloc((void **) &a, sizeof(int)*m*n, hipHostMallocDefault);
    hipHostMalloc((void **) &b, sizeof(int)*n*k, hipHostMallocDefault);

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = rand();
        }
    }

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            b[i * k + j] = rand();
        }
    }

    int *device_a, *device_b, *device_c;
    hipMalloc((void **) &device_a, sizeof(int)*m*n);
    hipMalloc((void **) &device_b, sizeof(int)*n*k);
    hipMalloc((void **) &device_c, sizeof(int)*m*k);

    hipMemcpy(device_a, a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    float elapsed_time_ms;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    matrix_mult<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, m, n, k);    
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, elapsed_time_ms);
    

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    hipHostFree(a);
    hipHostFree(b);
    return 0;
}